#include "hip/hip_runtime.h"
//
//                              VELOXCHEM
//         ----------------------------------------------------
//                     An Electronic Structure Code
//
//  Copyright © 2018-2023 by VeloxChem developers. All rights reserved.
//  Contact: https://veloxchem.org/contact
//
//  SPDX-License-Identifier: LGPL-3.0-or-later
//
//  This file is part of VeloxChem.
//
//  VeloxChem is free software: you can redistribute it and/or modify it under
//  the terms of the GNU Lesser General Public License as published by the Free
//  Software Foundation, either version 3 of the License, or (at your option)
//  any later version.
//
//  VeloxChem is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
//  License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License
//  along with VeloxChem. If not, see <https://www.gnu.org/licenses/>.

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <string>
#include <vector>

#include "DenseLinearAlgebra.hpp"
#include "DensityGridGenerator.hpp"
#include "DftFunc.hpp"
#include "DftSubMatrix.hpp"
#include "ErrorHandler.hpp"
#include "FunctionalParser.hpp"
#include "GtoFunc.hpp"
#include "GtoValues.hpp"
#include "GtoValuesRecF.hpp"
#include "MathFunc.hpp"
#include "MatrixFunc.hpp"
#include "MultiTimer.hpp"
#include "Prescreener.hpp"
#include "StringFormat.hpp"
#include "XCIntegratorGPU.hpp"

#define cudaSafe(e)                                                                                                       \
    {                                                                                                                     \
        hipError_t err = (e);                                                                                            \
        if (err != hipSuccess)                                                                                           \
        {                                                                                                                 \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                                                                      \
        }                                                                                                                 \
    }

#define cublasSafe(e)                                                                            \
    {                                                                                            \
        hipblasStatus_t err = (e);                                                                \
        if (err != HIPBLAS_STATUS_SUCCESS)                                                        \
        {                                                                                        \
            std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__ << ": " << std::endl; \
            std::exit(EXIT_FAILURE);                                                             \
        }                                                                                        \
    }

namespace gpu {  // gpu namespace

__global__ void
cudaLdaValuesDirectRecS(double*        gto_values,
                        const uint32_t row_offset,
                        const double*  gto_info,
                        const double*  grid_x,
                        const double*  grid_y,
                        const double*  grid_z,
                        const uint32_t grid_offset,
                        const uint32_t nrows,
                        const uint32_t npgtos,
                        const uint32_t ncols)
{
    const uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t g = blockDim.y * blockIdx.y + threadIdx.y;

    if ((i < nrows) && (g < ncols))
    {
        double s0 = 0.0;

        const auto g_x = grid_x[g + grid_offset];
        const auto g_y = grid_y[g + grid_offset];
        const auto g_z = grid_z[g + grid_offset];

        for (uint32_t j = 0; j < npgtos; j++)
        {
            const auto fexp  = gto_info[i + j * nrows + npgtos * nrows * 0];
            const auto fnorm = gto_info[i + j * nrows + npgtos * nrows * 1];
            const auto r_x   = gto_info[i + j * nrows + npgtos * nrows * 2];
            const auto r_y   = gto_info[i + j * nrows + npgtos * nrows * 3];
            const auto r_z   = gto_info[i + j * nrows + npgtos * nrows * 4];

            const auto gr_x = g_x - r_x;
            const auto gr_y = g_y - r_y;
            const auto gr_z = g_z - r_z;

            s0 += fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));
        }

        gto_values[g + (i + row_offset) * ncols] = s0;
    }
}

__global__ void
cudaLdaValuesDirectRecP(double*        gto_values_p3,
                        const uint32_t row_offset,
                        const double*  gto_info,
                        const double*  grid_x,
                        const double*  grid_y,
                        const double*  grid_z,
                        const uint32_t grid_offset,
                        const uint32_t nrows,
                        const uint32_t npgtos,
                        const uint32_t ncols)
{
    const uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t g = blockDim.y * blockIdx.y + threadIdx.y;

    if ((i < nrows) && (g < ncols))
    {
        double px = 0.0;
        double py = 0.0;
        double pz = 0.0;

        const auto g_x = grid_x[g + grid_offset];
        const auto g_y = grid_y[g + grid_offset];
        const auto g_z = grid_z[g + grid_offset];

        for (uint32_t j = 0; j < npgtos; j++)
        {
            const auto fexp  = gto_info[i + j * nrows + npgtos * nrows * 0];
            const auto fnorm = gto_info[i + j * nrows + npgtos * nrows * 1];
            const auto r_x   = gto_info[i + j * nrows + npgtos * nrows * 2];
            const auto r_y   = gto_info[i + j * nrows + npgtos * nrows * 3];
            const auto r_z   = gto_info[i + j * nrows + npgtos * nrows * 4];

            const auto gr_x = g_x - r_x;
            const auto gr_y = g_y - r_y;
            const auto gr_z = g_z - r_z;

            const auto fss = fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));

            px += gr_x * fss;
            py += gr_y * fss;
            pz += gr_z * fss;
        }

        // p-1: py
        // p_0: pz
        // p+1: px

        gto_values_p3[g + (i + row_offset) * ncols + nrows * ncols * 0] = py;
        gto_values_p3[g + (i + row_offset) * ncols + nrows * ncols * 1] = pz;
        gto_values_p3[g + (i + row_offset) * ncols + nrows * ncols * 2] = px;
    }
}

__global__ void
cudaLdaValuesDirectRecD(double*        gto_values_d5,
                        const uint32_t row_offset,
                        const double   f2_3,
                        const double*  gto_info,
                        const double*  grid_x,
                        const double*  grid_y,
                        const double*  grid_z,
                        const uint32_t grid_offset,
                        const uint32_t nrows,
                        const uint32_t npgtos,
                        const uint32_t ncols)
{
    const uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t g = blockDim.y * blockIdx.y + threadIdx.y;

    if ((i < nrows) && (g < ncols))
    {
        double dxx = 0.0;
        double dxy = 0.0;
        double dxz = 0.0;
        double dyy = 0.0;
        double dyz = 0.0;
        double dzz = 0.0;

        const auto g_x = grid_x[g + grid_offset];
        const auto g_y = grid_y[g + grid_offset];
        const auto g_z = grid_z[g + grid_offset];

        for (uint32_t j = 0; j < npgtos; j++)
        {
            const auto fexp  = gto_info[i + j * nrows + npgtos * nrows * 0];
            const auto fnorm = gto_info[i + j * nrows + npgtos * nrows * 1];
            const auto r_x   = gto_info[i + j * nrows + npgtos * nrows * 2];
            const auto r_y   = gto_info[i + j * nrows + npgtos * nrows * 3];
            const auto r_z   = gto_info[i + j * nrows + npgtos * nrows * 4];

            const auto gr_x = g_x - r_x;
            const auto gr_y = g_y - r_y;
            const auto gr_z = g_z - r_z;

            const auto fss = fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));

            dxx += gr_x * gr_x * fss;
            dxy += gr_x * gr_y * fss;
            dxz += gr_x * gr_z * fss;
            dyy += gr_y * gr_y * fss;
            dyz += gr_y * gr_z * fss;
            dzz += gr_z * gr_z * fss;
        }

        // d-2: dxy * f2_3
        // d-1: dyz * f2_3
        // d_0: dzz * 2.0 - dxx - dyy
        // d+1: dxz * f2_3
        // d+2: (dxx - dyy) * 0.5 * f2_3

        gto_values_d5[g + (i + row_offset) * ncols + nrows * ncols * 0] = dxy * f2_3;
        gto_values_d5[g + (i + row_offset) * ncols + nrows * ncols * 1] = dyz * f2_3;
        gto_values_d5[g + (i + row_offset) * ncols + nrows * ncols * 2] = (dzz * 2.0 - dxx - dyy);
        gto_values_d5[g + (i + row_offset) * ncols + nrows * ncols * 3] = dxz * f2_3;
        gto_values_d5[g + (i + row_offset) * ncols + nrows * ncols * 4] = (dxx - dyy) * 0.5 * f2_3;
    }
}

__global__ void
getSubDensityMatrix(double* d_den_mat, const double* d_den_mat_full, const uint32_t naos, const uint32_t* d_ao_inds, const uint32_t aocount)
{
    const uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t col = blockDim.y * blockIdx.y + threadIdx.y;

    if ((row < aocount) && (col < aocount))
    {
        const auto row_orig = d_ao_inds[row];
        const auto col_orig = d_ao_inds[col];

        d_den_mat[row * aocount + col] = d_den_mat_full[row_orig * naos + col_orig];
    }
}

__global__ void
zeroKohnShamMatrix(double* d_mat_Vxc_full, const uint32_t naos)
{
    const uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t col = blockDim.y * blockIdx.y + threadIdx.y;

    if ((row < naos) && (col < naos))
    {
        d_mat_Vxc_full[row * naos + col] = 0.0;
    }
}

__global__ void
distributeSubKohnShamMatrix(double* d_mat_Vxc_full, const uint32_t naos, const double* d_mat_Vxc, const uint32_t* d_ao_inds, const uint32_t aocount)
{
    const uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t col = blockDim.y * blockIdx.y + threadIdx.y;

    if ((row < aocount) && (col < aocount))
    {
        const auto row_orig = d_ao_inds[row];
        const auto col_orig = d_ao_inds[col];

        d_mat_Vxc_full[row_orig * naos + col_orig] += d_mat_Vxc[row * aocount + col];
    }
}

static auto
getGtoInfo(const CGtoBlock gto_block, const std::vector<int64_t>& gtos_mask) -> std::vector<double>
{
    // set up GTO values storage

    const auto nrows = mathfunc::countSignificantElements(gtos_mask);

    // set up GTOs data

    const auto gto_exps = gto_block.getExponents();

    const auto gto_norms = gto_block.getNormalizationFactors();

    const auto gto_coords = gto_block.getCoordinates();

    // set GTOs block dimensions

    const auto ncgtos = gto_block.getNumberOfBasisFunctions();

    const auto npgtos = gto_block.getNumberOfPrimitives();

    // set up data on host and device

    std::vector<double> gto_info(5 * nrows * npgtos);

    auto gto_info_ptr = gto_info.data();

    for (int64_t i = 0, irow = 0; i < ncgtos; i++)
    {
        if (gtos_mask[i] == 1)
        {
            const auto r_x = gto_coords[i][0];
            const auto r_y = gto_coords[i][1];
            const auto r_z = gto_coords[i][2];

            for (int64_t j = 0; j < npgtos; j++)
            {
                const auto fexp  = gto_exps[j * ncgtos + i];
                const auto fnorm = gto_norms[j * ncgtos + i];

                gto_info_ptr[irow + j * nrows + npgtos * nrows * 0] = fexp;
                gto_info_ptr[irow + j * nrows + npgtos * nrows * 1] = fnorm;
                gto_info_ptr[irow + j * nrows + npgtos * nrows * 2] = r_x;
                gto_info_ptr[irow + j * nrows + npgtos * nrows * 3] = r_y;
                gto_info_ptr[irow + j * nrows + npgtos * nrows * 4] = r_z;
            }

            irow++;
        }
    }

    return gto_info;
}

static auto
getGtoValuesForLdaDirect(double*                     d_gto_values,
                         const int64_t               row_offset,
                         double*                     d_gto_info,
                         const CGtoBlock&            gto_block,
                         const double*               d_grid_x,
                         const double*               d_grid_y,
                         const double*               d_grid_z,
                         const int64_t               grid_offset,
                         const int64_t               n_grid_points,
                         const std::vector<int64_t>& gtos_mask) -> void
{
    // number of useful CGTOs

    const auto nrows = mathfunc::countSignificantElements(gtos_mask);

    // number of primitives per CGTO

    const auto npgtos = gto_block.getNumberOfPrimitives();

    // number of grid points

    const auto ncols = n_grid_points;

    // prepare GTO information

    auto gto_info = gpu::getGtoInfo(gto_block, gtos_mask);

    cudaSafe(hipMemcpy(d_gto_info, gto_info.data(), gto_info.size() * sizeof(double), hipMemcpyHostToDevice));

    // evaluate GTO values on grid points

    dim3 threads_per_block(8, 32);

    dim3 num_blocks((nrows + threads_per_block.x - 1) / threads_per_block.x, (ncols + threads_per_block.y - 1) / threads_per_block.y);

    auto gto_ang = gto_block.getAngularMomentum();

    if (gto_ang == 0)
    {
        gpu::cudaLdaValuesDirectRecS<<<num_blocks, threads_per_block>>>(d_gto_values,
                                                                        static_cast<uint32_t>(row_offset),
                                                                        d_gto_info,
                                                                        d_grid_x,
                                                                        d_grid_y,
                                                                        d_grid_z,
                                                                        static_cast<uint32_t>(grid_offset),
                                                                        static_cast<uint32_t>(nrows),
                                                                        static_cast<uint32_t>(npgtos),
                                                                        static_cast<uint32_t>(ncols));
    }
    else if (gto_ang == 1)
    {
        gpu::cudaLdaValuesDirectRecP<<<num_blocks, threads_per_block>>>(d_gto_values,
                                                                        static_cast<uint32_t>(row_offset),
                                                                        d_gto_info,
                                                                        d_grid_x,
                                                                        d_grid_y,
                                                                        d_grid_z,
                                                                        static_cast<uint32_t>(grid_offset),
                                                                        static_cast<uint32_t>(nrows),
                                                                        static_cast<uint32_t>(npgtos),
                                                                        static_cast<uint32_t>(ncols));
    }
    else if (gto_ang == 2)
    {
        const double f2_3 = 2.0 * std::sqrt(3.0);

        gpu::cudaLdaValuesDirectRecD<<<num_blocks, threads_per_block>>>(d_gto_values,
                                                                        static_cast<uint32_t>(row_offset),
                                                                        f2_3,
                                                                        d_gto_info,
                                                                        d_grid_x,
                                                                        d_grid_y,
                                                                        d_grid_z,
                                                                        static_cast<uint32_t>(grid_offset),
                                                                        static_cast<uint32_t>(nrows),
                                                                        static_cast<uint32_t>(npgtos),
                                                                        static_cast<uint32_t>(ncols));
    }
    else
    {
        std::string err_ang("gpu::getGtoValuesForLdaDirect: Only implemented for s, p and d-orbitals");

        errors::assertMsgCritical(false, err_ang);
    }
}

auto
computeGtoValuesOnGridPoints(const CMolecule& molecule, const CMolecularBasis& basis, const CMolecularGrid& molecularGrid) -> CDenseMatrix
{
    // GTOs blocks and number of AOs

    const auto gto_blocks = gtofunc::makeGtoBlocks(basis, molecule);

    const auto naos = gtofunc::getNumberOfAtomicOrbitals(gto_blocks);

    int64_t max_ncgtos = 0, max_npgtos = 0;

    for (const auto& gto_block : gto_blocks)
    {
        const auto ncgtos = gto_block.getNumberOfBasisFunctions();
        const auto npgtos = gto_block.getNumberOfPrimitives();

        max_ncgtos = std::max(ncgtos, max_ncgtos);
        max_npgtos = std::max(npgtos, max_npgtos);
    }

    double* d_gto_info;

    cudaSafe(hipMalloc(&d_gto_info, 5 * max_ncgtos * max_npgtos * sizeof(double)));

    // GTO values on grid points

    CDenseMatrix allgtovalues(naos, molecularGrid.getNumberOfGridPoints());

    auto max_npoints_per_box = molecularGrid.getMaxNumberOfGridPointsPerBox();

    double* d_gaos;

    cudaSafe(hipMalloc(&d_gaos, naos * max_npoints_per_box * sizeof(double)));

    // coordinates of grid points

    auto xcoords = molecularGrid.getCoordinatesX();
    auto ycoords = molecularGrid.getCoordinatesY();
    auto zcoords = molecularGrid.getCoordinatesZ();

    auto n_total_grid_points = molecularGrid.getNumberOfGridPoints();

    double *d_grid_x, *d_grid_y, *d_grid_z;

    cudaSafe(hipMalloc(&d_grid_x, n_total_grid_points * sizeof(double)));
    cudaSafe(hipMalloc(&d_grid_y, n_total_grid_points * sizeof(double)));
    cudaSafe(hipMalloc(&d_grid_z, n_total_grid_points * sizeof(double)));

    cudaSafe(hipMemcpy(d_grid_x, xcoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(d_grid_y, ycoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(d_grid_z, zcoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));

    // counts and displacements of grid points in boxes

    auto counts = molecularGrid.getGridPointCounts();

    auto displacements = molecularGrid.getGridPointDisplacements();

    for (size_t box_id = 0; box_id < counts.size(); box_id++)
    {
        // grid points in box

        auto npoints = counts.data()[box_id];

        auto gridblockpos = displacements.data()[box_id];

        // dimension of grid box

        auto boxdim = prescr::getGridBoxDimension(gridblockpos, npoints, xcoords, ycoords, zcoords);

        // pre-screening

        std::vector<std::vector<int64_t>> cgto_mask_blocks, pre_ao_inds_blocks;

        std::vector<int64_t> aoinds;

        for (const auto& gto_block : gto_blocks)
        {
            // 0th order GTO derivative
            auto pre_scr_info = prescr::preScreenGtoBlock(gto_block, 0, 1.0e-12, boxdim);

            auto cgto_mask   = std::get<0>(pre_scr_info);
            auto pre_ao_inds = std::get<1>(pre_scr_info);

            cgto_mask_blocks.push_back(cgto_mask);

            pre_ao_inds_blocks.push_back(pre_ao_inds);

            for (const auto nu : pre_ao_inds)
            {
                aoinds.push_back(nu);
            }
        }

        const auto aocount = static_cast<int64_t>(aoinds.size());

        // GTO values on grid points

        CDenseMatrix mat_chi(aocount, npoints);

        const auto grid_x_ptr = xcoords + gridblockpos;
        const auto grid_y_ptr = ycoords + gridblockpos;
        const auto grid_z_ptr = zcoords + gridblockpos;

        std::vector<double> grid_x(grid_x_ptr, grid_x_ptr + npoints);
        std::vector<double> grid_y(grid_y_ptr, grid_y_ptr + npoints);
        std::vector<double> grid_z(grid_z_ptr, grid_z_ptr + npoints);

        // go through GTO blocks

        int64_t row_offset = 0;

        for (size_t i_block = 0; i_block < gto_blocks.size(); i_block++)
        {
            const auto& gto_block = gto_blocks[i_block];

            const auto& cgto_mask = cgto_mask_blocks[i_block];

            const auto& pre_ao_inds = pre_ao_inds_blocks[i_block];

            gpu::getGtoValuesForLdaDirect(d_gaos, row_offset, d_gto_info, gto_block, d_grid_x, d_grid_y, d_grid_z, gridblockpos, npoints, cgto_mask);

            row_offset += static_cast<int64_t>(pre_ao_inds.size());
        }

        cudaSafe(hipMemcpy(mat_chi.values(), d_gaos, aocount * npoints * sizeof(double), hipMemcpyDeviceToHost));

        for (int64_t nu = 0; nu < aocount; nu++)
        {
            std::memcpy(allgtovalues.row(aoinds[nu]) + gridblockpos, mat_chi.row(nu), npoints * sizeof(double));
        }
    }

    cudaSafe(hipFree(d_gto_info));
    cudaSafe(hipFree(d_gaos));
    cudaSafe(hipFree(d_grid_x));
    cudaSafe(hipFree(d_grid_y));
    cudaSafe(hipFree(d_grid_z));

    return allgtovalues;
}

__global__ void
cudaDensityOnGrids(double* d_rho, const double* d_mat_F, const double* d_gto_values, const uint32_t aocount, const uint32_t npoints)
{
    const uint32_t g = blockDim.x * blockIdx.x + threadIdx.x;

    if (g < npoints)
    {
        double rho_a = 0.0;

        for (uint32_t nu = 0; nu < aocount; nu++)
        {
            rho_a += d_mat_F[g + nu * npoints] * d_gto_values[g + nu * npoints];
        }

        d_rho[2 * g + 0] = rho_a;
        d_rho[2 * g + 1] = rho_a;
    }
}

static auto
generateDensityForLDA(double*                     rho,
                      double*                     d_rho,
                      double*                     d_mat_F,
                      double*                     d_den_mat,
                      uint32_t*                   d_ao_inds,
                      const double*               d_den_mat_full,
                      const int64_t               naos,
                      const std::vector<int64_t>& ao_inds,
                      const double*               d_gto_values,
                      const int64_t               npoints,
                      CMultiTimer&                timer) -> void
{
    timer.start("Density matrix slicing");

    const auto aocount = static_cast<int64_t>(ao_inds.size());

    std::vector<uint32_t> ao_inds_int32(aocount);

    for (int64_t ind = 0; ind < aocount; ind++)
    {
        ao_inds_int32[ind] = static_cast<uint32_t>(ao_inds[ind]);
    }

    cudaSafe(hipMemcpy(d_ao_inds, ao_inds_int32.data(), aocount * sizeof(uint32_t), hipMemcpyHostToDevice));

    dim3 threads_per_block(16, 16);

    dim3 num_blocks((aocount + threads_per_block.x - 1) / threads_per_block.x, (aocount + threads_per_block.y - 1) / threads_per_block.y);

    gpu::getSubDensityMatrix<<<num_blocks, threads_per_block>>>(
        d_den_mat, d_den_mat_full, static_cast<uint32_t>(naos), d_ao_inds, static_cast<uint32_t>(aocount));

    timer.stop("Density matrix slicing");

    timer.start("Density grid matmul");

    // density matrix: nao x nao
    auto narow = static_cast<uint32_t>(aocount);
    auto nacol = static_cast<uint32_t>(aocount);

    // GTO values: nao x npoints
    // auto nbrow = static_cast<uint32_t>(aocount);
    auto nbcol = static_cast<uint32_t>(npoints);

    // use cublas to get multAB(densityMatrix, gtoValues)

    hipblasHandle_t handle;
    cublasSafe(hipblasCreate(&handle));

    double alpha = 1.0, beta = 0.0;

    auto m = narow, k = nacol, n = nbcol;

    // we want row-major C = A * B but cublas is column-major.
    // so we do C^T = B^T * A^T instead.
    cublasSafe(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_gto_values, n, d_den_mat, k, &beta, d_mat_F, n));

    hipblasDestroy(handle);

    hipDeviceSynchronize();  // TODO: remove this

    timer.stop("Density grid matmul");

    timer.start("Density grid rho");

    threads_per_block = dim3(256);

    num_blocks = dim3((npoints + threads_per_block.x - 1) / threads_per_block.x);

    gpu::cudaDensityOnGrids<<<num_blocks, threads_per_block>>>(
        d_rho, d_mat_F, d_gto_values, static_cast<uint32_t>(aocount), static_cast<uint32_t>(npoints));

    cudaSafe(hipMemcpy(rho, d_rho, 2 * npoints * sizeof(double), hipMemcpyDeviceToHost));

    timer.stop("Density grid rho");
}

__global__ void
cudaGetMatrixG(double*        d_mat_G,
               const double*  d_grid_w,
               const uint32_t grid_offset,
               const uint32_t npoints,
               const double*  d_gto_values,
               const uint32_t aocount,
               const double*  d_vrho)
{
    const uint32_t g = blockDim.x * blockIdx.x + threadIdx.x;

    if (g < npoints)
    {
        for (int64_t nu = 0; nu < aocount; nu++)
        {
            d_mat_G[g + nu * npoints] = d_grid_w[g + grid_offset] * d_vrho[2 * g + 0] * d_gto_values[g + nu * npoints];
        }
    }
}

static auto
integratePartialVxcFockForLDA(double*         d_mat_G,
                              double*         d_mat_Vxc,
                              double*         d_mat_Vxc_full,
                              const int64_t   naos,
                              const double*   d_grid_w,
                              const int64_t   grid_offset,
                              const int64_t   npoints,
                              const uint32_t* d_ao_inds,
                              const double*   d_gto_values,
                              const int64_t   aocount,
                              const double*   d_vrho,
                              CMultiTimer&    timer) -> void
{
    timer.start("Vxc matrix G");

    dim3 threads_per_block(256);

    dim3 num_blocks((npoints + threads_per_block.x - 1) / threads_per_block.x);

    gpu::cudaGetMatrixG<<<num_blocks, threads_per_block>>>(
        d_mat_G, d_grid_w, static_cast<uint32_t>(grid_offset), static_cast<uint32_t>(npoints), d_gto_values, static_cast<uint32_t>(aocount), d_vrho);

    hipDeviceSynchronize();  // TODO: remove this

    timer.stop("Vxc matrix G");

    timer.start("Vxc matrix matmul");

    // GTO values: nao x npoints
    auto narow = static_cast<uint32_t>(aocount);
    auto nacol = static_cast<uint32_t>(npoints);

    // matrix G:   nao x npoints
    // matrix G^T: npoints x nao
    auto nbrow = static_cast<uint32_t>(aocount);
    // auto nbcol = static_cast<uint32_t>(npoints);

    auto m = narow, k = nacol, n = nbrow;

    hipblasHandle_t handle;
    cublasSafe(hipblasCreate(&handle));

    double alpha = 1.0, beta = 0.0;

    // TODO: double check transpose of d_mat_G

    // we want row-major C = A * B^T but cublas is column-major.
    // so we do C^T = B * A^T instead.
    cublasSafe(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, d_mat_G, k, d_gto_values, k, &beta, d_mat_Vxc, n));

    hipblasDestroy(handle);

    timer.stop("Vxc matrix matmul");

    timer.start("Vxc matrix dist.");

    threads_per_block = dim3(16, 16);

    num_blocks = dim3((aocount + threads_per_block.x - 1) / threads_per_block.x, (aocount + threads_per_block.y - 1) / threads_per_block.y);

    gpu::distributeSubKohnShamMatrix<<<num_blocks, threads_per_block>>>(
        d_mat_Vxc_full, static_cast<uint32_t>(naos), d_mat_Vxc, d_ao_inds, static_cast<uint32_t>(aocount));

    hipDeviceSynchronize();  // TODO: remove this

    timer.stop("Vxc matrix dist.");
}

static auto
integrateVxcFockForLDA(const CMolecule&        molecule,
                       const CMolecularBasis&  basis,
                       const CAODensityMatrix& densityMatrix,
                       const CMolecularGrid&   molecularGrid,
                       const CXCFunctional&    xcFunctional,
                       const std::string&      flag) -> CAOKohnShamMatrix
{
    CMultiTimer timer;

    timer.start("Total timing");

    timer.start("Preparation");

    // GTOs blocks and number of AOs

    const auto gto_blocks = gtofunc::makeGtoBlocks(basis, molecule);

    const auto naos = gtofunc::getNumberOfAtomicOrbitals(gto_blocks);

    std::string errnaos("gpu::integrateVxcFockForLDA: Inconsistent number of AOs");

    errors::assertMsgCritical((naos == densityMatrix.getNumberOfRows(0)) && (naos == densityMatrix.getNumberOfColumns(0)), errnaos);

    int64_t max_ncgtos = 0, max_npgtos = 0;

    for (const auto& gto_block : gto_blocks)
    {
        const auto ncgtos = gto_block.getNumberOfBasisFunctions();
        const auto npgtos = gto_block.getNumberOfPrimitives();

        max_ncgtos = std::max(ncgtos, max_ncgtos);
        max_npgtos = std::max(npgtos, max_npgtos);
    }

    double* d_gto_info;

    cudaSafe(hipMalloc(&d_gto_info, 5 * max_ncgtos * max_npgtos * sizeof(double)));

    uint32_t* d_ao_inds;

    cudaSafe(hipMalloc(&d_ao_inds, naos * sizeof(uint32_t)));

    // Kohn-Sham matrix

    bool closedshell = (fstr::upcase(flag) == std::string("CLOSEDSHELL"));

    CAOKohnShamMatrix mat_Vxc(naos, naos, closedshell);

    mat_Vxc.zero();

    // GTOs on grid points

    auto max_npoints_per_box = molecularGrid.getMaxNumberOfGridPointsPerBox();

    double *d_mat_Vxc_full, *d_den_mat_full, *d_den_mat, *d_gto_values, *d_mat_F;

    cudaSafe(hipMalloc(&d_den_mat, naos * naos * sizeof(double)));
    cudaSafe(hipMalloc(&d_gto_values, naos * max_npoints_per_box * sizeof(double)));
    cudaSafe(hipMalloc(&d_mat_F, naos * max_npoints_per_box * sizeof(double)));
    cudaSafe(hipMalloc(&d_den_mat_full, naos * naos * sizeof(double)));
    cudaSafe(hipMalloc(&d_mat_Vxc_full, naos * naos * sizeof(double)));

    cudaSafe(hipMemcpy(d_den_mat_full, densityMatrix.alphaDensity(0), naos * naos * sizeof(double), hipMemcpyHostToDevice));

    dim3 threads_per_block(16, 16);

    dim3 num_blocks((naos + threads_per_block.x - 1) / threads_per_block.x, (naos + threads_per_block.y - 1) / threads_per_block.y);

    gpu::zeroKohnShamMatrix<<<num_blocks, threads_per_block>>>(d_mat_Vxc_full, static_cast<uint32_t>(naos));

    // density and functional derivatives

    auto       ldafunc = xcFunctional.getFunctionalPointerToLdaComponent();
    const auto dim     = &(ldafunc->dim);

    std::vector<double> rho_data(dim->rho * max_npoints_per_box);

    std::vector<double> exc_data(dim->zk * max_npoints_per_box);
    std::vector<double> vrho_data(dim->vrho * max_npoints_per_box);

    auto rho = rho_data.data();

    auto exc  = exc_data.data();
    auto vrho = vrho_data.data();

    double *d_rho, *d_exc, *d_vrho;

    cudaSafe(hipMalloc(&d_rho, dim->rho * max_npoints_per_box * sizeof(double)));
    cudaSafe(hipMalloc(&d_exc, dim->zk * max_npoints_per_box * sizeof(double)));
    cudaSafe(hipMalloc(&d_vrho, dim->vrho * max_npoints_per_box * sizeof(double)));

    // initial values for XC energy and number of electrons

    double nele = 0.0, xcene = 0.0;

    // coordinates and weights of grid points

    auto xcoords = molecularGrid.getCoordinatesX();
    auto ycoords = molecularGrid.getCoordinatesY();
    auto zcoords = molecularGrid.getCoordinatesZ();

    auto weights = molecularGrid.getWeights();

    auto n_total_grid_points = molecularGrid.getNumberOfGridPoints();

    double *d_grid_x, *d_grid_y, *d_grid_z, *d_grid_w;

    cudaSafe(hipMalloc(&d_grid_x, n_total_grid_points * sizeof(double)));
    cudaSafe(hipMalloc(&d_grid_y, n_total_grid_points * sizeof(double)));
    cudaSafe(hipMalloc(&d_grid_z, n_total_grid_points * sizeof(double)));
    cudaSafe(hipMalloc(&d_grid_w, n_total_grid_points * sizeof(double)));

    cudaSafe(hipMemcpy(d_grid_x, xcoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(d_grid_y, ycoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(d_grid_z, zcoords, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(d_grid_w, weights, n_total_grid_points * sizeof(double), hipMemcpyHostToDevice));

    // counts and displacements of grid points in boxes

    auto counts = molecularGrid.getGridPointCounts();

    auto displacements = molecularGrid.getGridPointDisplacements();

    timer.stop("Preparation");

    for (size_t box_id = 0; box_id < counts.size(); box_id++)
    {
        // grid points in box

        auto npoints = counts.data()[box_id];

        auto gridblockpos = displacements.data()[box_id];

        // dimension of grid box

        auto boxdim = prescr::getGridBoxDimension(gridblockpos, npoints, xcoords, ycoords, zcoords);

        // prescreening

        timer.start("GTO pre-screening");

        std::vector<std::vector<int64_t>> cgto_mask_blocks, pre_ao_inds_blocks;

        std::vector<int64_t> aoinds;

        for (const auto& gto_block : gto_blocks)
        {
            // 0th order GTO derivative
            auto pre_scr_info = prescr::preScreenGtoBlock(gto_block, 0, 1.0e-12, boxdim);

            auto cgto_mask   = std::get<0>(pre_scr_info);
            auto pre_ao_inds = std::get<1>(pre_scr_info);

            cgto_mask_blocks.push_back(cgto_mask);

            pre_ao_inds_blocks.push_back(pre_ao_inds);

            for (const auto nu : pre_ao_inds)
            {
                aoinds.push_back(nu);
            }
        }

        const auto aocount = static_cast<int64_t>(aoinds.size());

        timer.stop("GTO pre-screening");

        if (aocount == 0) continue;

        // GTO values on grid points

        timer.start("GTO evaluation");

        int64_t row_offset = 0;

        for (size_t i_block = 0; i_block < gto_blocks.size(); i_block++)
        {
            const auto& gto_block = gto_blocks[i_block];

            const auto& cgto_mask = cgto_mask_blocks[i_block];

            const auto& pre_ao_inds = pre_ao_inds_blocks[i_block];

            gpu::getGtoValuesForLdaDirect(
                d_gto_values, row_offset, d_gto_info, gto_block, d_grid_x, d_grid_y, d_grid_z, gridblockpos, npoints, cgto_mask);

            row_offset += static_cast<int64_t>(pre_ao_inds.size());
        }

        hipDeviceSynchronize();  // TODO: remove this

        timer.stop("GTO evaluation");

        // generate sub density matrix and density grid

        if (closedshell)
        {
            gpu::generateDensityForLDA(rho, d_rho, d_mat_F, d_den_mat, d_ao_inds, d_den_mat_full, naos, aoinds, d_gto_values, npoints, timer);
        }
        else
        {
            // TODO: openshell
        }

        // compute exchange-correlation functional derivative

        timer.start("XC functional eval.");

        xcFunctional.compute_exc_vxc_for_lda(npoints, rho, exc, vrho);

        cudaSafe(hipMemcpy(d_exc, exc, dim->zk * npoints * sizeof(double), hipMemcpyHostToDevice));
        cudaSafe(hipMemcpy(d_vrho, vrho, dim->vrho * npoints * sizeof(double), hipMemcpyHostToDevice));

        timer.stop("XC functional eval.");

        // compute partial contribution to Vxc matrix and distribute partial
        // Vxc to full Kohn-Sham matrix

        if (closedshell)
        {
            // reuse d_den_mat and d_mat_F as working space

            auto d_mat_G   = d_mat_F;
            auto d_mat_Vxc = d_den_mat;

            gpu::integratePartialVxcFockForLDA(
                d_mat_G, d_mat_Vxc, d_mat_Vxc_full, naos, d_grid_w, gridblockpos, npoints, d_ao_inds, d_gto_values, aocount, d_vrho, timer);
        }
        else
        {
            // TODO: openshell
        }

        // compute partial contribution to XC energy

        timer.start("XC energy");

        for (int64_t g = 0; g < npoints; g++)
        {
            auto rho_total = rho[2 * g + 0] + rho[2 * g + 1];

            nele += weights[g + gridblockpos] * rho_total;

            xcene += weights[g + gridblockpos] * exc[g] * rho_total;
        }

        timer.stop("XC energy");
    }

    cudaSafe(hipMemcpy(mat_Vxc.getPointerToAlphaValues(), d_mat_Vxc_full, naos * naos * sizeof(double), hipMemcpyDeviceToHost));

    cudaSafe(hipFree(d_gto_info));
    cudaSafe(hipFree(d_ao_inds));

    cudaSafe(hipFree(d_den_mat));
    cudaSafe(hipFree(d_den_mat_full));
    cudaSafe(hipFree(d_gto_values));
    cudaSafe(hipFree(d_mat_F));

    cudaSafe(hipFree(d_rho));
    cudaSafe(hipFree(d_exc));
    cudaSafe(hipFree(d_vrho));

    cudaSafe(hipFree(d_grid_x));
    cudaSafe(hipFree(d_grid_y));
    cudaSafe(hipFree(d_grid_z));
    cudaSafe(hipFree(d_grid_w));

    mat_Vxc.setNumberOfElectrons(nele);

    mat_Vxc.setExchangeCorrelationEnergy(xcene);

    timer.stop("Total timing");

    std::cout << "\nTiming of GPU integrator\n";
    std::cout << "------------------------\n";
    std::cout << timer.getSummary() << std::endl;

    return mat_Vxc;
}

auto
integrateVxcFock(const CMolecule&        molecule,
                 const CMolecularBasis&  basis,
                 const CAODensityMatrix& densityMatrix,
                 const CMolecularGrid&   molecularGrid,
                 const std::string&      xcFuncLabel) -> CAOKohnShamMatrix
{
    auto fvxc = vxcfuncs::getExchangeCorrelationFunctional(xcFuncLabel);

    auto xcfuntype = fvxc.getFunctionalType();

    auto flag = densityMatrix.isClosedShell() ? std::string("CLOSEDSHELL") : std::string("OPENSHELL");

    std::string erropenshell("gpu::integrateVxcFock: Only implemented for closed-shell");

    errors::assertMsgCritical(densityMatrix.isClosedShell(), erropenshell);

    if (xcfuntype == xcfun::lda) return gpu::integrateVxcFockForLDA(molecule, basis, densityMatrix, molecularGrid, fvxc, flag);

    /*
    if (xcfuntype == xcfun::gga) return gpu::integrateVxcFockForGGA(molecule, basis, densityMatrix, molecularGrid, fvxc, flag);

    if (xcfuntype == xcfun::mgga) return gpu::integrateVxcFockForMGGA(molecule, basis, densityMatrix, molecularGrid, fvxc, flag);

    std::string errxcfuntype("gpu::integrateVxcFock: Only implemented for LDA/GGA/meta-GGA");

    errors::assertMsgCritical(false, errxcfuntype);
    */

    return CAOKohnShamMatrix();
}

}  // namespace gpu
