#include "hip/hip_runtime.h"
//
//                              VELOXCHEM
//         ----------------------------------------------------
//                     An Electronic Structure Code
//
//  Copyright © 2018-2023 by VeloxChem developers. All rights reserved.
//  Contact: https://veloxchem.org/contact
//
//  SPDX-License-Identifier: LGPL-3.0-or-later
//
//  This file is part of VeloxChem.
//
//  VeloxChem is free software: you can redistribute it and/or modify it under
//  the terms of the GNU Lesser General Public License as published by the Free
//  Software Foundation, either version 3 of the License, or (at your option)
//  any later version.
//
//  VeloxChem is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
//  License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License
//  along with VeloxChem. If not, see <https://www.gnu.org/licenses/>.

#include <omp.h>

#include <cstdint>
#include <cstring>
#include <iostream>
#include <string>
#include <vector>

#include "DftFunc.hpp"
#include "GtoFunc.hpp"
#include "GtoValues.hpp"
#include "GtoValuesRecF.hpp"
#include "MathFunc.hpp"
#include "MatrixFunc.hpp"
#include "Prescreener.hpp"
#include "XCIntegratorGPU.hpp"

// clang-format off
#define cudaSafe(err) { gpu::cudaCheck(err); }
// clang-format on

namespace gpu {  // gpu namespace

inline void
cudaCheck(hipError_t err)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void
cudaLdaValuesRecS(double*        gto_values,
                  const double*  gto_info,
                  const double*  grid_xyz,
                  const uint32_t nrows,
                  const uint32_t npgtos,
                  const uint32_t ncols)
{
    const uint32_t g = blockDim.x * blockIdx.x + threadIdx.x;

    if (g < ncols)
    {
        const auto g_x = grid_xyz[g];
        const auto g_y = grid_xyz[g + ncols];
        const auto g_z = grid_xyz[g + ncols * 2];

        for (uint32_t i = 0, idx = 0; i < nrows; i++)
        {
            gto_values[g + i * ncols] = 0.0;

            for (uint32_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_info[idx * 5 + 0];
                const auto fnorm = gto_info[idx * 5 + 1];
                const auto r_x   = gto_info[idx * 5 + 2];
                const auto r_y   = gto_info[idx * 5 + 3];
                const auto r_z   = gto_info[idx * 5 + 4];

                const auto gr_x = g_x - r_x;
                const auto gr_y = g_y - r_y;
                const auto gr_z = g_z - r_z;

                gto_values[g + i * ncols] += fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));
            }
        }
    }
}

__global__ void
cudaLdaValuesRecP(double*        gto_values_p3,
                  const double*  gto_info,
                  const double*  grid_xyz,
                  const uint32_t nrows,
                  const uint32_t npgtos,
                  const uint32_t ncols)
{
    const uint32_t g = blockDim.x * blockIdx.x + threadIdx.x;

    if (g < ncols)
    {
        const auto g_x = grid_xyz[g];
        const auto g_y = grid_xyz[g + ncols];
        const auto g_z = grid_xyz[g + ncols * 2];

        for (uint32_t i = 0, idx = 0; i < nrows; i++)
        {
            gto_values_p3[g + i * ncols]                     = 0.0;
            gto_values_p3[g + i * ncols + nrows * ncols]     = 0.0;
            gto_values_p3[g + i * ncols + nrows * ncols * 2] = 0.0;

            for (uint32_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_info[idx * 5 + 0];
                const auto fnorm = gto_info[idx * 5 + 1];
                const auto r_x   = gto_info[idx * 5 + 2];
                const auto r_y   = gto_info[idx * 5 + 3];
                const auto r_z   = gto_info[idx * 5 + 4];

                const auto gr_x = g_x - r_x;
                const auto gr_y = g_y - r_y;
                const auto gr_z = g_z - r_z;

                const auto fss = fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));

                gto_values_p3[g + i * ncols] += gr_x * fss;
                gto_values_p3[g + i * ncols + nrows * ncols] += gr_y * fss;
                gto_values_p3[g + i * ncols + nrows * ncols * 2] += gr_z * fss;
            }
        }
    }
}

__global__ void
cudaLdaValuesRecD(double*        gto_values_d6,
                  const double*  gto_info,
                  const double*  grid_xyz,
                  const uint32_t nrows,
                  const uint32_t npgtos,
                  const uint32_t ncols)
{
    const uint32_t g = blockDim.x * blockIdx.x + threadIdx.x;

    if (g < ncols)
    {
        const auto g_x = grid_xyz[g];
        const auto g_y = grid_xyz[g + ncols];
        const auto g_z = grid_xyz[g + ncols * 2];

        for (uint32_t i = 0, idx = 0; i < nrows; i++)
        {
            gto_values_d6[g + i * ncols]                     = 0.0;
            gto_values_d6[g + i * ncols + nrows * ncols]     = 0.0;
            gto_values_d6[g + i * ncols + nrows * ncols * 2] = 0.0;
            gto_values_d6[g + i * ncols + nrows * ncols * 3] = 0.0;
            gto_values_d6[g + i * ncols + nrows * ncols * 4] = 0.0;
            gto_values_d6[g + i * ncols + nrows * ncols * 5] = 0.0;

            for (uint32_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_info[idx * 5 + 0];
                const auto fnorm = gto_info[idx * 5 + 1];
                const auto r_x   = gto_info[idx * 5 + 2];
                const auto r_y   = gto_info[idx * 5 + 3];
                const auto r_z   = gto_info[idx * 5 + 4];

                const auto gr_x = g_x - r_x;
                const auto gr_y = g_y - r_y;
                const auto gr_z = g_z - r_z;

                const auto fss = fnorm * std::exp(-fexp * (gr_x * gr_x + gr_y * gr_y + gr_z * gr_z));

                gto_values_d6[g + i * ncols] += gr_x * gr_x * fss;
                gto_values_d6[g + i * ncols + nrows * ncols] += gr_x * gr_y * fss;
                gto_values_d6[g + i * ncols + nrows * ncols * 2] += gr_x * gr_z * fss;
                gto_values_d6[g + i * ncols + nrows * ncols * 3] += gr_y * gr_y * fss;
                gto_values_d6[g + i * ncols + nrows * ncols * 4] += gr_y * gr_z * fss;
                gto_values_d6[g + i * ncols + nrows * ncols * 5] += gr_z * gr_z * fss;
            }
        }
    }
}

auto
getLdaValuesRecS(const CGtoBlock&            gto_block,
                 const std::vector<double>&  grid_coords_x,
                 const std::vector<double>&  grid_coords_y,
                 const std::vector<double>&  grid_coords_z,
                 const std::vector<int64_t>& gtos_mask) -> CMatrix
{
    // set up GTO values storage

    const auto nrows = mathfunc::countSignificantElements(gtos_mask);

    const auto ncols = static_cast<int64_t>(grid_coords_x.size());

    auto gto_values = matfunc::makeMatrix("LDA", nrows, ncols);

    auto submat = gto_values.getSubMatrix({0, 0});

    submat->zero();

    // set up GTOs data

    const auto gto_exps = gto_block.getExponents();

    const auto gto_norms = gto_block.getNormalizationFactors();

    const auto gto_coords = gto_block.getCoordinates();

    // set up grid data

    auto g_x = grid_coords_x.data();

    auto g_y = grid_coords_y.data();

    auto g_z = grid_coords_z.data();

    // set GTOs block dimensions

    const auto ncgtos = gto_block.getNumberOfBasisFunctions();

    const auto npgtos = gto_block.getNumberOfPrimitives();

    // set up data on host and device

    double *gto_values_h, *gto_info_h, *grid_xyz_h;

    cudaSafe(hipHostMalloc(&gto_values_h, nrows * ncols * sizeof(double)));
    cudaSafe(hipHostMalloc(&gto_info_h, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipHostMalloc(&grid_xyz_h, 3 * ncols * sizeof(double)));

    double *gto_values_d, *gto_info_d, *grid_xyz_d;

    cudaSafe(hipMalloc(&gto_values_d, nrows * ncols * sizeof(double)));
    cudaSafe(hipMalloc(&gto_info_d, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipMalloc(&grid_xyz_d, 3 * ncols * sizeof(double)));

    for (int64_t i = 0, idx = 0; i < ncgtos; i++)
    {
        if (gtos_mask[i] == 1)
        {
            const auto r_x = gto_coords[i][0];
            const auto r_y = gto_coords[i][1];
            const auto r_z = gto_coords[i][2];

            for (int64_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_exps[j * ncgtos + i];
                const auto fnorm = gto_norms[j * ncgtos + i];

                gto_info_h[idx * 5 + 0] = fexp;
                gto_info_h[idx * 5 + 1] = fnorm;
                gto_info_h[idx * 5 + 2] = r_x;
                gto_info_h[idx * 5 + 3] = r_y;
                gto_info_h[idx * 5 + 4] = r_z;
            }
        }
    }

    for (int64_t k = 0; k < ncols; k++)
    {
        grid_xyz_h[k]             = g_x[k];
        grid_xyz_h[k + ncols]     = g_y[k];
        grid_xyz_h[k + ncols * 2] = g_z[k];
    }

    cudaSafe(hipMemcpy(gto_info_d, gto_info_h, 5 * nrows * npgtos * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(grid_xyz_d, grid_xyz_h, 3 * ncols * sizeof(double), hipMemcpyHostToDevice));

    dim3 threads_per_block(256);
    dim3 nblocks((ncols + 255) / 256);

    gpu::cudaLdaValuesRecS<<<nblocks, threads_per_block>>>(
        gto_values_d, gto_info_d, grid_xyz_d, static_cast<uint32_t>(nrows), static_cast<uint32_t>(npgtos), static_cast<uint32_t>(ncols));

    cudaSafe(hipMemcpy(gto_values_h, gto_values_d, nrows * ncols * sizeof(double), hipMemcpyDeviceToHost));

    for (int64_t irow = 0; irow < nrows; irow++)
    {
        for (int64_t k = 0; k < ncols; k++)
        {
            submat->at(irow, k, false) += gto_values_h[k + irow * ncols];
        }
    }

    cudaSafe(hipHostFree(gto_values_h));
    cudaSafe(hipHostFree(gto_info_h));
    cudaSafe(hipHostFree(grid_xyz_h));

    cudaSafe(hipFree(gto_values_d));
    cudaSafe(hipFree(gto_info_d));
    cudaSafe(hipFree(grid_xyz_d));

    return gto_values;
}

auto
getLdaValuesRecP(const CGtoBlock&            gto_block,
                 const std::vector<double>&  grid_coords_x,
                 const std::vector<double>&  grid_coords_y,
                 const std::vector<double>&  grid_coords_z,
                 const std::vector<int64_t>& gtos_mask) -> CMatrix
{
    // set up GTO values storage

    const auto nrows = mathfunc::countSignificantElements(gtos_mask);

    const auto ncols = static_cast<int64_t>(grid_coords_x.size());

    auto gto_values = matfunc::makeMatrix("LDA", 3 * nrows, ncols);

    auto submat = gto_values.getSubMatrix({0, 0});

    submat->zero();

    // set up GTOs data

    const auto gto_exps = gto_block.getExponents();

    const auto gto_norms = gto_block.getNormalizationFactors();

    const auto gto_coords = gto_block.getCoordinates();

    // set up grid data

    auto g_x = grid_coords_x.data();

    auto g_y = grid_coords_y.data();

    auto g_z = grid_coords_z.data();

    // set GTOs block dimensions

    const auto ncgtos = gto_block.getNumberOfBasisFunctions();

    const auto npgtos = gto_block.getNumberOfPrimitives();

    // set up data on host and device

    double *gto_values_h, *gto_info_h, *grid_xyz_h;

    cudaSafe(hipHostMalloc(&gto_values_h, 3 * nrows * ncols * sizeof(double)));
    cudaSafe(hipHostMalloc(&gto_info_h, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipHostMalloc(&grid_xyz_h, 3 * ncols * sizeof(double)));

    double *gto_values_d, *gto_info_d, *grid_xyz_d;

    cudaSafe(hipMalloc(&gto_values_d, 3 * nrows * ncols * sizeof(double)));
    cudaSafe(hipMalloc(&gto_info_d, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipMalloc(&grid_xyz_d, 3 * ncols * sizeof(double)));

    for (int64_t i = 0, idx = 0; i < ncgtos; i++)
    {
        if (gtos_mask[i] == 1)
        {
            const auto r_x = gto_coords[i][0];
            const auto r_y = gto_coords[i][1];
            const auto r_z = gto_coords[i][2];

            for (int64_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_exps[j * ncgtos + i];
                const auto fnorm = gto_norms[j * ncgtos + i];

                gto_info_h[idx * 5 + 0] = fexp;
                gto_info_h[idx * 5 + 1] = fnorm;
                gto_info_h[idx * 5 + 2] = r_x;
                gto_info_h[idx * 5 + 3] = r_y;
                gto_info_h[idx * 5 + 4] = r_z;
            }
        }
    }

    for (int64_t k = 0; k < ncols; k++)
    {
        grid_xyz_h[k]             = g_x[k];
        grid_xyz_h[k + ncols]     = g_y[k];
        grid_xyz_h[k + ncols * 2] = g_z[k];
    }

    cudaSafe(hipMemcpy(gto_info_d, gto_info_h, 5 * nrows * npgtos * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(grid_xyz_d, grid_xyz_h, 3 * ncols * sizeof(double), hipMemcpyHostToDevice));

    dim3 threads_per_block(256);
    dim3 nblocks((ncols + 255) / 256);

    gpu::cudaLdaValuesRecP<<<nblocks, threads_per_block>>>(
        gto_values_d, gto_info_d, grid_xyz_d, static_cast<uint32_t>(nrows), static_cast<uint32_t>(npgtos), static_cast<uint32_t>(ncols));

    cudaSafe(hipMemcpy(gto_values_h, gto_values_d, 3 * nrows * ncols * sizeof(double), hipMemcpyDeviceToHost));

    for (int64_t irow = 0; irow < nrows; irow++)
    {
        for (int64_t k = 0; k < ncols; k++)
        {
            // buffer_x, 2 * nrows + irow
            // buffer_y,             irow
            // buffer_z,     nrows + irow

            submat->at(irow + nrows * 2, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 0];
            submat->at(irow + nrows * 0, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 1];
            submat->at(irow + nrows * 1, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 2];
        }
    }

    cudaSafe(hipHostFree(gto_values_h));
    cudaSafe(hipHostFree(gto_info_h));
    cudaSafe(hipHostFree(grid_xyz_h));

    cudaSafe(hipFree(gto_values_d));
    cudaSafe(hipFree(gto_info_d));
    cudaSafe(hipFree(grid_xyz_d));

    return gto_values;
}

auto
getLdaValuesRecD(const CGtoBlock&            gto_block,
                 const std::vector<double>&  grid_coords_x,
                 const std::vector<double>&  grid_coords_y,
                 const std::vector<double>&  grid_coords_z,
                 const std::vector<int64_t>& gtos_mask) -> CMatrix
{
    // spherical transformation factors

    const double f2_3 = 2.0 * std::sqrt(3.0);

    // set up GTO values storage

    const auto nrows = mathfunc::countSignificantElements(gtos_mask);

    const auto ncols = static_cast<int64_t>(grid_coords_x.size());

    auto gto_values = matfunc::makeMatrix("LDA", 5 * nrows, ncols);

    auto submat = gto_values.getSubMatrix({0, 0});

    submat->zero();

    // set up GTOs data

    const auto gto_exps = gto_block.getExponents();

    const auto gto_norms = gto_block.getNormalizationFactors();

    const auto gto_coords = gto_block.getCoordinates();

    // set up grid data

    auto g_x = grid_coords_x.data();

    auto g_y = grid_coords_y.data();

    auto g_z = grid_coords_z.data();

    // set GTOs block dimensions

    const auto ncgtos = gto_block.getNumberOfBasisFunctions();

    const auto npgtos = gto_block.getNumberOfPrimitives();

    // set up data on host and device

    double *gto_values_h, *gto_info_h, *grid_xyz_h;

    cudaSafe(hipHostMalloc(&gto_values_h, 6 * nrows * ncols * sizeof(double)));
    cudaSafe(hipHostMalloc(&gto_info_h, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipHostMalloc(&grid_xyz_h, 3 * ncols * sizeof(double)));

    double *gto_values_d, *gto_info_d, *grid_xyz_d;

    cudaSafe(hipMalloc(&gto_values_d, 6 * nrows * ncols * sizeof(double)));
    cudaSafe(hipMalloc(&gto_info_d, 5 * nrows * npgtos * sizeof(double)));
    cudaSafe(hipMalloc(&grid_xyz_d, 3 * ncols * sizeof(double)));

    for (int64_t i = 0, idx = 0; i < ncgtos; i++)
    {
        if (gtos_mask[i] == 1)
        {
            const auto r_x = gto_coords[i][0];
            const auto r_y = gto_coords[i][1];
            const auto r_z = gto_coords[i][2];

            for (int64_t j = 0; j < npgtos; j++, idx++)
            {
                const auto fexp  = gto_exps[j * ncgtos + i];
                const auto fnorm = gto_norms[j * ncgtos + i];

                gto_info_h[idx * 5 + 0] = fexp;
                gto_info_h[idx * 5 + 1] = fnorm;
                gto_info_h[idx * 5 + 2] = r_x;
                gto_info_h[idx * 5 + 3] = r_y;
                gto_info_h[idx * 5 + 4] = r_z;
            }
        }
    }

    for (int64_t k = 0; k < ncols; k++)
    {
        grid_xyz_h[k]             = g_x[k];
        grid_xyz_h[k + ncols]     = g_y[k];
        grid_xyz_h[k + ncols * 2] = g_z[k];
    }

    cudaSafe(hipMemcpy(gto_info_d, gto_info_h, 5 * nrows * npgtos * sizeof(double), hipMemcpyHostToDevice));
    cudaSafe(hipMemcpy(grid_xyz_d, grid_xyz_h, 3 * ncols * sizeof(double), hipMemcpyHostToDevice));

    dim3 threads_per_block(256);
    dim3 nblocks((ncols + 255) / 256);

    gpu::cudaLdaValuesRecD<<<nblocks, threads_per_block>>>(
        gto_values_d, gto_info_d, grid_xyz_d, static_cast<uint32_t>(nrows), static_cast<uint32_t>(npgtos), static_cast<uint32_t>(ncols));

    cudaSafe(hipMemcpy(gto_values_h, gto_values_d, 6 * nrows * ncols * sizeof(double), hipMemcpyDeviceToHost));

    for (int64_t irow = 0; irow < nrows; irow++)
    {
        for (int64_t k = 0; k < ncols; k++)
        {
            // buffer_xx, -1.0,        2 * nrows + irow
            // buffer_xx, 0.5 * f2_3,  4 * nrows + irow
            // buffer_xy, f2_3,                    irow
            // buffer_xz, f2_3,        3 * nrows + irow
            // buffer_yy, -1.0,        2 * nrows + irow
            // buffer_yy, -0.5 * f2_3, 4 * nrows + irow
            // buffer_yz, f2_3,            nrows + irow
            // buffer_zz, 2.0,         2 * nrows + irow

            submat->at(irow + nrows * 2, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 0] * (-1.0);
            submat->at(irow + nrows * 4, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 0] * 0.5 * f2_3;
            submat->at(irow + nrows * 0, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 1] * f2_3;
            submat->at(irow + nrows * 3, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 2] * f2_3;
            submat->at(irow + nrows * 2, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 3] * (-1.0);
            submat->at(irow + nrows * 4, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 3] * (-0.5) * f2_3;
            submat->at(irow + nrows * 1, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 4] * f2_3;
            submat->at(irow + nrows * 2, k, false) += gto_values_h[k + irow * ncols + nrows * ncols * 5] * 2.0;
        }
    }

    cudaSafe(hipHostFree(gto_values_h));
    cudaSafe(hipHostFree(gto_info_h));
    cudaSafe(hipHostFree(grid_xyz_h));

    cudaSafe(hipFree(gto_values_d));
    cudaSafe(hipFree(gto_info_d));
    cudaSafe(hipFree(grid_xyz_d));

    return gto_values;
}

auto
getGtoValuesForLda(const CGtoBlock&            gto_block,
                   const std::vector<double>&  grid_coords_x,
                   const std::vector<double>&  grid_coords_y,
                   const std::vector<double>&  grid_coords_z,
                   const std::vector<int64_t>& gtos_mask) -> CMatrix
{
    auto gto_ang = gto_block.getAngularMomentum();

    if (gto_ang == 0)
    {
        return gpu::getLdaValuesRecS(gto_block, grid_coords_x, grid_coords_y, grid_coords_z, gtos_mask);
    }
    else if (gto_ang == 1)
    {
        return gpu::getLdaValuesRecP(gto_block, grid_coords_x, grid_coords_y, grid_coords_z, gtos_mask);
    }
    else if (gto_ang == 2)
    {
        return gpu::getLdaValuesRecD(gto_block, grid_coords_x, grid_coords_y, grid_coords_z, gtos_mask);
    }
    else if (gto_ang == 3)
    {
        return gtoval::getLdaValuesRecF(gto_block, grid_coords_x, grid_coords_y, grid_coords_z, gtos_mask);
    }

    return CMatrix();
}

auto
computeGtoValuesOnGridPoints(const CMolecule& molecule, const CMolecularBasis& basis, const CMolecularGrid& molecularGrid) -> CDenseMatrix
{
    // number of OpenMP threads

    auto nthreads = omp_get_max_threads();

    // GTOs blocks and number of AOs

    const auto gto_blocks = gtofunc::makeGtoBlocks(basis, molecule);

    const auto naos = gtofunc::getNumberOfAtomicOrbitals(gto_blocks);

    // GTO values on grid points

    CDenseMatrix allgtovalues(naos, molecularGrid.getNumberOfGridPoints());

    // coordinates and weights of grid points

    auto xcoords = molecularGrid.getCoordinatesX();
    auto ycoords = molecularGrid.getCoordinatesY();
    auto zcoords = molecularGrid.getCoordinatesZ();

    // counts and displacements of grid points in boxes

    auto counts = molecularGrid.getGridPointCounts();

    auto displacements = molecularGrid.getGridPointDisplacements();

    for (size_t box_id = 0; box_id < counts.size(); box_id++)
    {
        // grid points in box

        auto npoints = counts.data()[box_id];

        auto gridblockpos = displacements.data()[box_id];

        // dimension of grid box

        auto boxdim = prescr::getGridBoxDimension(gridblockpos, npoints, xcoords, ycoords, zcoords);

        // compute GTO values on grid points

        const auto grid_x_ptr = xcoords + gridblockpos;
        const auto grid_y_ptr = ycoords + gridblockpos;
        const auto grid_z_ptr = zcoords + gridblockpos;

        std::vector<double> grid_x(grid_x_ptr, grid_x_ptr + npoints);
        std::vector<double> grid_y(grid_y_ptr, grid_y_ptr + npoints);
        std::vector<double> grid_z(grid_z_ptr, grid_z_ptr + npoints);

        // go through GTO blocks

        for (const auto& gto_block : gto_blocks)
        {
            // prescreen GTO block

            // 0th order GTO derivative
            auto pre_scr_info = prescr::preScreenGtoBlock(gto_block, 0, 1.0e-12, boxdim);

            auto cgto_mask   = std::get<0>(pre_scr_info);
            auto pre_ao_inds = std::get<1>(pre_scr_info);

            // GTO values on grid points

            auto cmat = gpu::getGtoValuesForLda(gto_block, grid_x, grid_y, grid_z, cgto_mask);

            auto submat_ptr = cmat.getSubMatrix({0, 0});

            auto subgaos_ptr = submat_ptr->getData();

            for (int64_t nu = 0; nu < static_cast<int64_t>(pre_ao_inds.size()); nu++)
            {
                std::memcpy(allgtovalues.row(pre_ao_inds[nu]) + gridblockpos, subgaos_ptr + nu * npoints, npoints * sizeof(double));
            }
        }
    }

    return allgtovalues;
}

}  // namespace gpu
