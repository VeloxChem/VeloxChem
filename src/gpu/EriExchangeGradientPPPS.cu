#include "hip/hip_runtime.h"
//
//                                   VELOXCHEM
//              ----------------------------------------------------
//                          An Electronic Structure Code
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  Copyright 2018-2025 VeloxChem developers
//
//  Redistribution and use in source and binary forms, with or without modification,
//  are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this
//     list of conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software without
//     specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
//  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
//  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//  DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
//  HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT
//  OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "GpuRuntime.hpp"


#include "BoysFuncGPU.hpp"
#include "EriExchangeGradientPPPS.hpp"

namespace gpu {  // gpu namespace

__global__ void __launch_bounds__(TILE_SIZE_K)
computeExchangeGradientPPPS_I_0(double*         grad_x,
                                const uint32_t  grad_cart_ind,
                                const double    frac_exact_exchange,
                                const uint32_t* pair_inds_i_for_K_pp,
                                const uint32_t* pair_inds_k_for_K_pp,
                                const double*   D_ik_for_K_pp,
                                const uint32_t  pair_inds_count_for_K_pp,
                                const double*   s_prim_info,
                                const uint32_t* s_prim_aoinds,
                                const uint32_t  s_prim_count,
                                const double*   p_prim_info,
                                const uint32_t* p_prim_aoinds,
                                const uint32_t  p_prim_count,
                                const double    ps_max_D,
                                const double*   mat_D_full_AO,
                                const uint32_t  naos,
                                const double*   Q_K_pp,
                                const double*   Q_K_ps,
                                const uint32_t* D_inds_K_pp,
                                const uint32_t* D_inds_K_ps,
                                const uint32_t* pair_displs_K_pp,
                                const uint32_t* pair_displs_K_ps,
                                const uint32_t* pair_counts_K_pp,
                                const uint32_t* pair_counts_K_ps,
                                const double*   pair_data_K_pp,
                                const double*   pair_data_K_ps,
                                const uint32_t* prim_cart_ao_to_atom_inds,
                                const double*   boys_func_table,
                                const double*   boys_func_ft,
                                const double    omega,
                                const double    eri_threshold)
{
    // each thread block scans over [i?|k?] and sum up to a primitive K matrix element
    // J. Chem. Theory Comput. 2009, 5, 4, 1004-1015

    __shared__ double   ERIs[TILE_DIM_Y_K][TILE_DIM_X_K + 1];
    __shared__ uint32_t i, k, count_i, count_k, displ_i, displ_k;
    __shared__ double   a_i, r_i[3], a_k, r_k[3], ik_factor_D;
    __shared__ uint32_t a0, c0;
    __shared__ double   delta[3][3];

    const uint32_t ik = blockIdx.x;

    double d2 = 1.0;

    ERIs[threadIdx.y][threadIdx.x] = 0.0;

    if ((threadIdx.y == 0) && (threadIdx.x == 0))
    {
        count_i = 0;
        count_k = 0;

        delta[0][0] = 1.0; delta[0][1] = 0.0; delta[0][2] = 0.0;
        delta[1][0] = 0.0; delta[1][1] = 1.0; delta[1][2] = 0.0;
        delta[2][0] = 0.0; delta[2][1] = 0.0; delta[2][2] = 1.0;

        if (ik < pair_inds_count_for_K_pp)
        {
            i = pair_inds_i_for_K_pp[ik];
            k = pair_inds_k_for_K_pp[ik];

            count_i = pair_counts_K_pp[i];
            count_k = pair_counts_K_ps[k];

            displ_i = pair_displs_K_pp[i];
            displ_k = pair_displs_K_ps[k];

            a_i = p_prim_info[i / 3 + p_prim_count * 0];

            r_i[0] = p_prim_info[i / 3 + p_prim_count * 2];
            r_i[1] = p_prim_info[i / 3 + p_prim_count * 3];
            r_i[2] = p_prim_info[i / 3 + p_prim_count * 4];

            a_k = p_prim_info[k / 3 + p_prim_count * 0];

            r_k[0] = p_prim_info[k / 3 + p_prim_count * 2];
            r_k[1] = p_prim_info[k / 3 + p_prim_count * 3];
            r_k[2] = p_prim_info[k / 3 + p_prim_count * 4];

            ik_factor_D = (static_cast<double>(i != k) + 1.0) * D_ik_for_K_pp[ik];

            a0 = i % 3;
            c0 = k % 3;

        }
    }

    __syncthreads();

    for (uint32_t m = 0; m < (count_i + TILE_DIM_Y_K - 1) / TILE_DIM_Y_K; m++)
    {
        const uint32_t j = m * TILE_DIM_Y_K + threadIdx.y;

        // sync threads before starting a new scan
        __syncthreads();

        double Q_ij, a_j, r_j[3], S_ij_00, S1, inv_S1;
        double PA_0, PB_0, PA_x;
        uint32_t j_prim, j_cgto, b0;

        if ((ik < pair_inds_count_for_K_pp) && (j < count_i))
        {
            Q_ij   = Q_K_pp[displ_i + j];

            j_prim = D_inds_K_pp[displ_i + j];

            j_cgto = p_prim_aoinds[(j_prim / 3) + p_prim_count * (j_prim % 3)];

            a_j = p_prim_info[j_prim / 3 + p_prim_count * 0];

            r_j[0] = p_prim_info[j_prim / 3 + p_prim_count * 2];
            r_j[1] = p_prim_info[j_prim / 3 + p_prim_count * 3];
            r_j[2] = p_prim_info[j_prim / 3 + p_prim_count * 4];

            S1 = a_i + a_j;
            inv_S1 = 1.0 / S1;

            S_ij_00 = pair_data_K_pp[displ_i + j];

            PA_x = (a_j * inv_S1) * (r_j[grad_cart_ind] - r_i[grad_cart_ind]);

            b0 = j_prim % 3;

            PA_0 = (a_j  * inv_S1) * (r_j[a0] - r_i[a0]);
            PB_0 = (-a_i * inv_S1) * (r_j[b0] - r_i[b0]);

        }


        for (uint32_t n = 0; n < (count_k + TILE_DIM_X_K - 1) / TILE_DIM_X_K; n++)
        {
            const uint32_t l = n * TILE_DIM_X_K + threadIdx.x;

            if ((ik >= pair_inds_count_for_K_pp) || (j >= count_i) || (l >= count_k) || (fabs(Q_ij * Q_K_ps[displ_k + l] * ps_max_D) <= eri_threshold))
            {
                break;
            }

            const auto l_prim = D_inds_K_ps[displ_k + l];

            const auto l_cgto = s_prim_aoinds[l_prim];

            const auto a_l = s_prim_info[l_prim + s_prim_count * 0];

            const double r_l[3] = {s_prim_info[l_prim + s_prim_count * 2],
                                   s_prim_info[l_prim + s_prim_count * 3],
                                   s_prim_info[l_prim + s_prim_count * 4]};

            const auto S_kl_00 = pair_data_K_ps[displ_k + l];


            // J. Chem. Phys. 84, 3963-3974 (1986)

            const auto S2 = a_k + a_l;

            const auto inv_S2 = 1.0 / S2;
            const auto inv_S4 = 1.0 / (S1 + S2);

            const double PQ[3] = {(a_k * r_k[0] + a_l * r_l[0]) * inv_S2 - (a_i * r_i[0] + a_j * r_j[0]) * inv_S1,
                                  (a_k * r_k[1] + a_l * r_l[1]) * inv_S2 - (a_i * r_i[1] + a_j * r_j[1]) * inv_S1,
                                  (a_k * r_k[2] + a_l * r_l[2]) * inv_S2 - (a_i * r_i[2] + a_j * r_j[2]) * inv_S1};

            const auto r2_PQ = PQ[0] * PQ[0] + PQ[1] * PQ[1] + PQ[2] * PQ[2];

            const auto rho = S1 * S2 * inv_S4;

            if (omega != 0.0) d2 = omega * omega / (rho + omega * omega);

            const auto Lambda = sqrt(4.0 * rho * d2 * MATH_CONST_INV_PI);

            double F4_t[5];

            gpu::computeBoysFunction(F4_t, rho * d2 * r2_PQ, 4, boys_func_table, boys_func_ft);

            if (omega != 0.0)
            {
                F4_t[1] *= d2;
                F4_t[2] *= d2 * d2;
                F4_t[3] *= d2 * d2 * d2;
                F4_t[4] *= d2 * d2 * d2 * d2;
            }

            const double QC_x = (a_l * inv_S2) * (r_l[grad_cart_ind] - r_k[grad_cart_ind]);

            const auto QC_0 = (a_l * inv_S2) * (r_l[c0] - r_k[c0]);


            // mu grad

            const double eri_ijkl = Lambda * S_ij_00 * S_kl_00 * (

                            F4_t[0] * inv_S1 * a_i * (
                                delta[a0][grad_cart_ind] * (PB_0 * QC_0)
                                + delta[b0][grad_cart_ind] * (PA_0 * QC_0)
                                + delta[a0][b0] * (PA_x * QC_0)
                            )

                            + F4_t[0] * 2.0 * a_i * (
                                + PA_0 * PA_x * PB_0 * QC_0
                            )

                            + F4_t[0] * (-1.0) * (
                                delta[a0][grad_cart_ind] * (PB_0 * QC_0)
                            )

                            + F4_t[1] * S2 * inv_S1 * inv_S4 * a_i * (
                                delta[b0][grad_cart_ind] * (PA_0 * QC_0 * (-1.0) + PQ[a0] * QC_0)
                                + delta[a0][grad_cart_ind] * (PB_0 * QC_0 * (-1.0) + PQ[b0] * QC_0)
                                + delta[a0][b0] * (PA_x * QC_0 * (-1.0) + PQ[grad_cart_ind] * QC_0)
                            )

                            + F4_t[1] * inv_S4 * a_i * (
                                delta[c0][grad_cart_ind] * (PA_0 * PB_0)
                                + delta[b0][grad_cart_ind] * (PA_0 * PQ[c0] * (-1.0))
                                + delta[a0][c0] * (PA_x * PB_0)
                                + delta[a0][grad_cart_ind] * (PB_0 * PQ[c0] * (-1.0))
                                + delta[a0][b0] * (PA_x * PQ[c0] * (-1.0))
                                + delta[b0][c0] * (PA_0 * PA_x)
                            )

                            + F4_t[1] * (-0.5) * inv_S4 * (
                                delta[a0][grad_cart_ind] * delta[b0][c0]
                            )

                            + F4_t[1] * 2.0 * S1 * inv_S4 * a_i * (
                                + PA_0 * PA_x * PB_0 * PQ[c0] * (-1.0)
                            )

                            + F4_t[1] * S1 * inv_S4 * (
                                delta[a0][grad_cart_ind] * (PB_0 * PQ[c0])
                            )

                            + F4_t[1] * 2.0 * S2 * inv_S4 * a_i * (
                                + PA_0 * PA_x * PQ[b0] * QC_0
                                + PA_0 * PB_0 * PQ[grad_cart_ind] * QC_0
                                + PA_x * PB_0 * PQ[a0] * QC_0
                            )

                            + F4_t[1] * (-1.0) * S2 * inv_S4 * (
                                delta[a0][grad_cart_ind] * (PQ[b0] * QC_0)
                            )

                            + F4_t[1] * 0.5 * inv_S1 * inv_S4 * a_i * (
                                (delta[a0][b0] * delta[c0][grad_cart_ind] + delta[a0][c0] * delta[b0][grad_cart_ind] + delta[a0][grad_cart_ind] * delta[b0][c0])
                            )

                            + F4_t[2] * (-0.5) * S2 * inv_S1 * inv_S4 * inv_S4 * a_i * (
                                (delta[a0][b0] * delta[c0][grad_cart_ind] + delta[a0][c0] * delta[b0][grad_cart_ind] + delta[a0][grad_cart_ind] * delta[b0][c0])
                            )

                            + F4_t[2] * (-1.0) * S2 * S2 * inv_S1 * inv_S4 * inv_S4 * a_i * (
                                delta[b0][grad_cart_ind] * (PQ[a0] * QC_0)
                                + delta[a0][grad_cart_ind] * (PQ[b0] * QC_0)
                                + delta[a0][b0] * (PQ[grad_cart_ind] * QC_0)
                            )

                            + F4_t[2] * S2 * inv_S4 * inv_S4 * a_i * (
                                delta[c0][grad_cart_ind] * (PA_0 * PQ[b0] + PB_0 * PQ[a0])
                                + delta[a0][grad_cart_ind] * (PQ[b0] * PQ[c0] * (-1.0) + PB_0 * PQ[c0])
                                + delta[a0][c0] * (PA_x * PQ[b0] + PB_0 * PQ[grad_cart_ind])
                                + delta[b0][grad_cart_ind] * (PQ[a0] * PQ[c0] * (-1.0) + PA_0 * PQ[c0])
                                + delta[a0][b0] * (PQ[c0] * PQ[grad_cart_ind] * (-1.0) + PA_x * PQ[c0])
                                + delta[b0][c0] * (PA_0 * PQ[grad_cart_ind] + PA_x * PQ[a0])
                            )

                            + F4_t[2] * 2.0 * S1 * S2 * inv_S4 * inv_S4 * a_i * (
                                + PA_0 * PA_x * PQ[b0] * PQ[c0] * (-1.0)
                                + PA_0 * PB_0 * PQ[c0] * PQ[grad_cart_ind] * (-1.0)
                                + PA_x * PB_0 * PQ[a0] * PQ[c0] * (-1.0)
                            )

                            + F4_t[2] * 2.0 * S2 * S2 * inv_S4 * inv_S4 * a_i * (
                                + PA_0 * PQ[b0] * PQ[grad_cart_ind] * QC_0
                                + PA_x * PQ[a0] * PQ[b0] * QC_0
                                + PB_0 * PQ[a0] * PQ[grad_cart_ind] * QC_0
                            )

                            + F4_t[2] * S1 * S2 * inv_S4 * inv_S4 * (
                                delta[a0][grad_cart_ind] * (PQ[b0] * PQ[c0])
                            )

                            + F4_t[3] * (-2.0) * S1 * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * a_i * (
                                PA_0 * PQ[b0] * PQ[c0] * PQ[grad_cart_ind]
                                + PA_x * PQ[a0] * PQ[b0] * PQ[c0]
                                + PB_0 * PQ[a0] * PQ[c0] * PQ[grad_cart_ind]
                            )

                            + F4_t[3] * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * a_i * (
                                delta[c0][grad_cart_ind] * (PQ[a0] * PQ[b0])
                                + delta[b0][grad_cart_ind] * (PQ[a0] * PQ[c0])
                                + delta[b0][c0] * (PQ[a0] * PQ[grad_cart_ind])
                                + delta[a0][grad_cart_ind] * (PQ[b0] * PQ[c0])
                                + delta[a0][c0] * (PQ[b0] * PQ[grad_cart_ind])
                                + delta[a0][b0] * (PQ[c0] * PQ[grad_cart_ind])
                            )

                            + F4_t[3] * 2.0 * S2 * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * a_i * (
                                PQ[a0] * PQ[b0] * PQ[grad_cart_ind] * QC_0
                            )

                            + F4_t[4] * (-2.0) * S1 * S2 * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * inv_S4 * a_i * (
                                PQ[a0] * PQ[b0] * PQ[c0] * PQ[grad_cart_ind]
                            )

                    );

            ERIs[threadIdx.y][threadIdx.x] -= eri_ijkl * mat_D_full_AO[j_cgto * naos + l_cgto];

        }
    }


    __syncthreads();

    if ((threadIdx.y == 0) && (threadIdx.x == 0) && (ik < pair_inds_count_for_K_pp))
    {
        double grad_i_x = 0.0;

        for (uint32_t y = 0; y < TILE_DIM_Y_K; y++)
        {
            for (uint32_t x = 0; x < TILE_DIM_X_K; x++)
            {
                grad_i_x += ERIs[y][x];
            }
        }

        atomicAdd(grad_x + prim_cart_ao_to_atom_inds[s_prim_count + i], grad_i_x * ik_factor_D * 2.0 * frac_exact_exchange);
    }
}

__global__ void __launch_bounds__(TILE_SIZE_K)
computeExchangeGradientPPPS_K_0(double*         grad_x,
                                const uint32_t  grad_cart_ind,
                                const double    frac_exact_exchange,
                                const uint32_t* pair_inds_i_for_K_pp,
                                const uint32_t* pair_inds_k_for_K_pp,
                                const double*   D_ik_for_K_pp,
                                const uint32_t  pair_inds_count_for_K_pp,
                                const double*   s_prim_info,
                                const uint32_t* s_prim_aoinds,
                                const uint32_t  s_prim_count,
                                const double*   p_prim_info,
                                const uint32_t* p_prim_aoinds,
                                const uint32_t  p_prim_count,
                                const double    ps_max_D,
                                const double*   mat_D_full_AO,
                                const uint32_t  naos,
                                const double*   Q_K_pp,
                                const double*   Q_K_ps,
                                const uint32_t* D_inds_K_pp,
                                const uint32_t* D_inds_K_ps,
                                const uint32_t* pair_displs_K_pp,
                                const uint32_t* pair_displs_K_ps,
                                const uint32_t* pair_counts_K_pp,
                                const uint32_t* pair_counts_K_ps,
                                const double*   pair_data_K_pp,
                                const double*   pair_data_K_ps,
                                const uint32_t* prim_cart_ao_to_atom_inds,
                                const double*   boys_func_table,
                                const double*   boys_func_ft,
                                const double    omega,
                                const double    eri_threshold)
{
    // each thread block scans over [i?|k?] and sum up to a primitive K matrix element
    // J. Chem. Theory Comput. 2009, 5, 4, 1004-1015

    __shared__ double   ERIs[TILE_DIM_Y_K][TILE_DIM_X_K + 1];
    __shared__ uint32_t i, k, count_i, count_k, displ_i, displ_k;
    __shared__ double   a_i, r_i[3], a_k, r_k[3], ik_factor_D;
    __shared__ uint32_t a0, c0;
    __shared__ double   delta[3][3];

    const uint32_t ik = blockIdx.x;

    double d2 = 1.0;

    ERIs[threadIdx.y][threadIdx.x] = 0.0;

    if ((threadIdx.y == 0) && (threadIdx.x == 0))
    {
        count_i = 0;
        count_k = 0;

        delta[0][0] = 1.0; delta[0][1] = 0.0; delta[0][2] = 0.0;
        delta[1][0] = 0.0; delta[1][1] = 1.0; delta[1][2] = 0.0;
        delta[2][0] = 0.0; delta[2][1] = 0.0; delta[2][2] = 1.0;

        if (ik < pair_inds_count_for_K_pp)
        {
            i = pair_inds_i_for_K_pp[ik];
            k = pair_inds_k_for_K_pp[ik];

            count_i = pair_counts_K_pp[i];
            count_k = pair_counts_K_ps[k];

            displ_i = pair_displs_K_pp[i];
            displ_k = pair_displs_K_ps[k];

            a_i = p_prim_info[i / 3 + p_prim_count * 0];

            r_i[0] = p_prim_info[i / 3 + p_prim_count * 2];
            r_i[1] = p_prim_info[i / 3 + p_prim_count * 3];
            r_i[2] = p_prim_info[i / 3 + p_prim_count * 4];

            a_k = p_prim_info[k / 3 + p_prim_count * 0];

            r_k[0] = p_prim_info[k / 3 + p_prim_count * 2];
            r_k[1] = p_prim_info[k / 3 + p_prim_count * 3];
            r_k[2] = p_prim_info[k / 3 + p_prim_count * 4];

            ik_factor_D = (static_cast<double>(i != k) + 1.0) * D_ik_for_K_pp[ik];

            a0 = i % 3;
            c0 = k % 3;

        }
    }

    __syncthreads();

    for (uint32_t m = 0; m < (count_i + TILE_DIM_Y_K - 1) / TILE_DIM_Y_K; m++)
    {
        const uint32_t j = m * TILE_DIM_Y_K + threadIdx.y;

        // sync threads before starting a new scan
        __syncthreads();

        double Q_ij, a_j, r_j[3], S_ij_00, S1, inv_S1;
        double PA_0, PB_0, PA_x;
        uint32_t j_prim, j_cgto, b0;

        if ((ik < pair_inds_count_for_K_pp) && (j < count_i))
        {
            Q_ij   = Q_K_pp[displ_i + j];

            j_prim = D_inds_K_pp[displ_i + j];

            j_cgto = p_prim_aoinds[(j_prim / 3) + p_prim_count * (j_prim % 3)];

            a_j = p_prim_info[j_prim / 3 + p_prim_count * 0];

            r_j[0] = p_prim_info[j_prim / 3 + p_prim_count * 2];
            r_j[1] = p_prim_info[j_prim / 3 + p_prim_count * 3];
            r_j[2] = p_prim_info[j_prim / 3 + p_prim_count * 4];

            S1 = a_i + a_j;
            inv_S1 = 1.0 / S1;

            S_ij_00 = pair_data_K_pp[displ_i + j];

            PA_x = (a_j * inv_S1) * (r_j[grad_cart_ind] - r_i[grad_cart_ind]);

            b0 = j_prim % 3;

            PA_0 = (a_j  * inv_S1) * (r_j[a0] - r_i[a0]);
            PB_0 = (-a_i * inv_S1) * (r_j[b0] - r_i[b0]);

        }


        for (uint32_t n = 0; n < (count_k + TILE_DIM_X_K - 1) / TILE_DIM_X_K; n++)
        {
            const uint32_t l = n * TILE_DIM_X_K + threadIdx.x;

            if ((ik >= pair_inds_count_for_K_pp) || (j >= count_i) || (l >= count_k) || (fabs(Q_ij * Q_K_ps[displ_k + l] * ps_max_D) <= eri_threshold))
            {
                break;
            }

            const auto l_prim = D_inds_K_ps[displ_k + l];

            const auto l_cgto = s_prim_aoinds[l_prim];

            const auto a_l = s_prim_info[l_prim + s_prim_count * 0];

            const double r_l[3] = {s_prim_info[l_prim + s_prim_count * 2],
                                   s_prim_info[l_prim + s_prim_count * 3],
                                   s_prim_info[l_prim + s_prim_count * 4]};

            const auto S_kl_00 = pair_data_K_ps[displ_k + l];


            // J. Chem. Phys. 84, 3963-3974 (1986)

            const auto S2 = a_k + a_l;

            const auto inv_S2 = 1.0 / S2;
            const auto inv_S4 = 1.0 / (S1 + S2);

            const double PQ[3] = {(a_k * r_k[0] + a_l * r_l[0]) * inv_S2 - (a_i * r_i[0] + a_j * r_j[0]) * inv_S1,
                                  (a_k * r_k[1] + a_l * r_l[1]) * inv_S2 - (a_i * r_i[1] + a_j * r_j[1]) * inv_S1,
                                  (a_k * r_k[2] + a_l * r_l[2]) * inv_S2 - (a_i * r_i[2] + a_j * r_j[2]) * inv_S1};

            const auto r2_PQ = PQ[0] * PQ[0] + PQ[1] * PQ[1] + PQ[2] * PQ[2];

            const auto rho = S1 * S2 * inv_S4;

            if (omega != 0.0) d2 = omega * omega / (rho + omega * omega);

            const auto Lambda = sqrt(4.0 * rho * d2 * MATH_CONST_INV_PI);

            double F4_t[5];

            gpu::computeBoysFunction(F4_t, rho * d2 * r2_PQ, 4, boys_func_table, boys_func_ft);

            if (omega != 0.0)
            {
                F4_t[1] *= d2;
                F4_t[2] *= d2 * d2;
                F4_t[3] *= d2 * d2 * d2;
                F4_t[4] *= d2 * d2 * d2 * d2;
            }

            const double QC_x = (a_l * inv_S2) * (r_l[grad_cart_ind] - r_k[grad_cart_ind]);

            const auto QC_0 = (a_l * inv_S2) * (r_l[c0] - r_k[c0]);


            // lambda grad

            const double eri_ijkl = Lambda * S_ij_00 * S_kl_00 * (

                            F4_t[0] * (-0.5) * inv_S1 * (
                                delta[a0][b0] * delta[c0][grad_cart_ind]
                            )

                            + F4_t[0] * inv_S2 * a_k * (
                                delta[c0][grad_cart_ind] * (PA_0 * PB_0)
                            )

                            + F4_t[0] * 2.0 * a_k * (
                                + PA_0 * PB_0 * QC_0 * QC_x
                            )

                            + F4_t[0] * (-1.0) * (
                                delta[c0][grad_cart_ind] * (PA_0 * PB_0)
                            )

                            + F4_t[0] * 0.5 * inv_S1 * inv_S2 * a_k * (
                                delta[a0][b0] * delta[c0][grad_cart_ind]
                            )

                            + F4_t[0] * inv_S1 * a_k * (
                                delta[a0][b0] * (QC_0 * QC_x)
                            )

                            + F4_t[1] * (-0.5) * inv_S1 * inv_S4 * a_k * (
                                delta[a0][b0] * delta[c0][grad_cart_ind]
                            )

                            + F4_t[1] * (-0.5) * inv_S2 * inv_S4 * a_k * (
                                delta[a0][b0] * delta[c0][grad_cart_ind]
                            )

                            + F4_t[1] * (-1.0) * S1 * inv_S2 * inv_S4 * a_k * (
                                delta[c0][grad_cart_ind] * (PA_0 * PB_0)
                            )

                            + F4_t[1] * (-1.0) * S2 * inv_S1 * inv_S4 * a_k * (
                                delta[a0][b0] * (QC_0 * QC_x)
                            )

                            + F4_t[1] * inv_S4 * a_k * (
                                delta[c0][grad_cart_ind] * (PA_0 * PQ[b0] + PB_0 * PQ[a0])
                                + delta[a0][grad_cart_ind] * (PB_0 * QC_0)
                                + delta[a0][c0] * (PB_0 * QC_x)
                                + delta[a0][b0] * (PQ[c0] * QC_x * (-1.0) + PQ[grad_cart_ind] * QC_0 * (-1.0))
                                + delta[b0][grad_cart_ind] * (PA_0 * QC_0)
                                + delta[b0][c0] * (PA_0 * QC_x)
                            )

                            + F4_t[1] * 2.0 * S1 * inv_S4 * a_k * (
                                + PA_0 * PB_0 * PQ[c0] * QC_x * (-1.0)
                                + PA_0 * PB_0 * PQ[grad_cart_ind] * QC_0 * (-1.0)
                            )

                            + F4_t[1] * 2.0 * S2 * inv_S4 * a_k * (
                                + PA_0 * PQ[b0] * QC_0 * QC_x
                                + PB_0 * PQ[a0] * QC_0 * QC_x
                            )

                            + F4_t[1] * (-1.0) * S2 * inv_S4 * (
                                delta[c0][grad_cart_ind] * (PA_0 * PQ[b0] + PB_0 * PQ[a0])
                            )

                            + F4_t[1] * 0.5 * S2 * inv_S1 * inv_S4 * (
                                delta[a0][b0] * delta[c0][grad_cart_ind]
                            )

                            + F4_t[2] * S1 * inv_S4 * inv_S4 * a_k * (
                                delta[c0][grad_cart_ind] * (PA_0 * PQ[b0] * (-1.0) + PB_0 * PQ[a0] * (-1.0))
                                + delta[b0][grad_cart_ind] * (PA_0 * PQ[c0] * (-1.0))
                                + delta[b0][c0] * (PA_0 * PQ[grad_cart_ind] * (-1.0))
                                + delta[a0][grad_cart_ind] * (PB_0 * PQ[c0] * (-1.0))
                                + delta[a0][c0] * (PB_0 * PQ[grad_cart_ind] * (-1.0))
                                + delta[a0][b0] * (PQ[c0] * PQ[grad_cart_ind])
                            )

                            + F4_t[2] * 2.0 * S1 * S1 * inv_S4 * inv_S4 * a_k * (
                                + PA_0 * PB_0 * PQ[c0] * PQ[grad_cart_ind]
                            )

                            + F4_t[2] * (-2.0) * S1 * S2 * inv_S4 * inv_S4 * a_k * (
                                PA_0 * PQ[b0] * PQ[c0] * QC_x
                                + PA_0 * PQ[b0] * PQ[grad_cart_ind] * QC_0
                                + PB_0 * PQ[a0] * PQ[c0] * QC_x
                                + PB_0 * PQ[a0] * PQ[grad_cart_ind] * QC_0
                            )

                            + F4_t[2] * (-1.0) * S2 * S2 * inv_S4 * inv_S4 * (
                                delta[c0][grad_cart_ind] * (PQ[a0] * PQ[b0])
                            )

                            + F4_t[2] * 0.5 * inv_S4 * inv_S4 * a_k * (
                                (delta[a0][b0] * delta[c0][grad_cart_ind] + delta[a0][c0] * delta[b0][grad_cart_ind] + delta[a0][grad_cart_ind] * delta[b0][c0])
                            )

                            + F4_t[2] * S2 * inv_S4 * inv_S4 * a_k * (
                                delta[c0][grad_cart_ind] * (PQ[a0] * PQ[b0])
                                + delta[b0][grad_cart_ind] * (PQ[a0] * QC_0)
                                + delta[b0][c0] * (PQ[a0] * QC_x)
                                + delta[a0][grad_cart_ind] * (PQ[b0] * QC_0)
                                + delta[a0][c0] * (PQ[b0] * QC_x)
                                + delta[a0][b0] * (PQ[c0] * QC_x + PQ[grad_cart_ind] * QC_0)
                            )

                            + F4_t[2] * 2.0 * S2 * S2 * inv_S4 * inv_S4 * a_k * (
                                PQ[a0] * PQ[b0] * QC_0 * QC_x
                            )

                            + F4_t[3] * (-1.0) * S1 * S2 * inv_S4 * inv_S4 * inv_S4 * a_k * (
                                delta[c0][grad_cart_ind] * (PQ[a0] * PQ[b0])
                                + delta[b0][grad_cart_ind] * (PQ[a0] * PQ[c0])
                                + delta[b0][c0] * (PQ[a0] * PQ[grad_cart_ind])
                                + delta[a0][grad_cart_ind] * (PQ[b0] * PQ[c0])
                                + delta[a0][c0] * (PQ[b0] * PQ[grad_cart_ind])
                                + delta[a0][b0] * (PQ[c0] * PQ[grad_cart_ind])
                            )

                            + F4_t[3] * 2.0 * S1 * S1 * S2 * inv_S4 * inv_S4 * inv_S4 * a_k * (
                                + PA_0 * PQ[b0] * PQ[c0] * PQ[grad_cart_ind]
                                + PB_0 * PQ[a0] * PQ[c0] * PQ[grad_cart_ind]
                            )

                            + F4_t[3] * (-2.0) * S1 * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * a_k * (
                                PQ[a0] * PQ[b0] * PQ[c0] * QC_x
                                + PQ[a0] * PQ[b0] * PQ[grad_cart_ind] * QC_0
                            )

                            + F4_t[4] * 2.0 * S1 * S1 * S2 * S2 * inv_S4 * inv_S4 * inv_S4 * inv_S4 * a_k * (
                                PQ[a0] * PQ[b0] * PQ[c0] * PQ[grad_cart_ind]
                            )


                    );

            ERIs[threadIdx.y][threadIdx.x] -= eri_ijkl * mat_D_full_AO[l_cgto * naos + j_cgto];
        }
    }


    __syncthreads();

    if ((threadIdx.y == 0) && (threadIdx.x == 0) && (ik < pair_inds_count_for_K_pp))
    {
        double grad_k_x = 0.0;

        for (uint32_t y = 0; y < TILE_DIM_Y_K; y++)
        {
            for (uint32_t x = 0; x < TILE_DIM_X_K; x++)
            {
                grad_k_x += ERIs[y][x];
            }
        }

        atomicAdd(grad_x + prim_cart_ao_to_atom_inds[s_prim_count + k], grad_k_x * ik_factor_D * 2.0 * frac_exact_exchange);
    }
}


}  // namespace gpu
